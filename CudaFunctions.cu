#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "math.h"
#include "Utilities.h"

// ======================================= Service methods =======================================
__host__ void allocateImageOnGPU(Image image, int **deviceImage) 
{
	int colorsInImage = image.dimension * image.dimension;
	
	hipError_t error = hipSuccess;
		
	// Allocates and copies the object to GPU
	error = hipMalloc(deviceImage, colorsInImage * sizeof(int));
	if (error != hipSuccess)
	{
		printf("Cannot allocate GPU memory for image: %s (%d)\n", hipGetErrorString(error), error);
    		exit(0);
  	}
  	error = hipMemcpy(*deviceImage, image.colorsMatrix, colorsInImage * sizeof(int), hipMemcpyHostToDevice);
	if (error != hipSuccess)
	{
		printf("Cannot copy image to GPU: %s (%d)\n", hipGetErrorString(error), error);
    		exit(0);
  	}
}
__host__ void freeImageFromGPU(int **deviceImage) 
{
	hipError_t error = hipSuccess;
	
	// Frees the picture from GPU memory
	error = hipFree(*deviceImage);
	if (error != hipSuccess)
	{
		printf("Cannot free image from GPU: %s (%d)\n", hipGetErrorString(error), error);
    		exit(0);
  	}
}

// ======================================= Device methods =======================================
__device__ __host__ int getPositionsPerDimension(int pictureDimension, int objectDimension) // called from both CPU and GPU
{
	return (pictureDimension - objectDimension) + 1;
}

__device__ int getPictureOffset(int matchingOffset, int objectOffset, int pictureDimension, int objectDimension)
{
	int positionsPerDimension = getPositionsPerDimension(pictureDimension, objectDimension);
	int matchingRow = matchingOffset / positionsPerDimension, matchingColumn = matchingOffset % positionsPerDimension;
	int objectRow = objectOffset / objectDimension, objectColumn = objectOffset % objectDimension;
	
	return ((matchingRow + objectRow) * pictureDimension) + (matchingColumn + objectColumn);
}

__device__ float difference(int p, int o) 
{
	return abs(((float)p - o) / p);
}

// ======================================= Kernel methods =======================================
__global__ void searchPositions(int pictureDimension, int *devicePictureColorsMatrix,
	int objectDimension, int *deviceObjectColorsMatrix, float *deviceMatchingsArray,
	int *devicePositionFlagsArray, float matchingEpsilon) 
{
	// MODUS OPERANDI:
	// 1. Given MxM picture P and NxN object O (N <= M), there is a submatrix of (M - N + 1)x(M - N + 1) possible positions to find O within P.
	// 2. Each possible position requires NxN calculations for the search, thus (M - N + 1)xN threads are required per dimension.
	// 3. Since the threads are allocated in blocks of 1024 threads (32 threads per block dimension), some threads might be allocated but never used.
	// 4. If a thread is necessary for the search, it's ID in relation to the "required threads submatrix" can be calculated, and from it the
	//    matching ID (offset in matchings array) and offset within O.
	// 5. After extracting row and column from matching offset and object offset, picture offset is ((matchingRow + objectRow) * M) + (matchingColumn + objectColumn).

	int positionsPerDimension = getPositionsPerDimension(pictureDimension, objectDimension);
	int threadsPerDimension = positionsPerDimension * objectDimension;
	int isNecessaryThread = (threadIdx.x < threadsPerDimension) && (threadIdx.y < threadsPerDimension);
	int threadID, pictureOffset, objectOffset, matchingOffset;
	
	// Checks threads position in relation to the required amount of threads
	if (isNecessaryThread)
	{
		// Initializes required variables
		threadID = (threadIdx.y * threadsPerDimension) + threadIdx.x;
		objectOffset = threadID % (objectDimension * objectDimension);
		matchingOffset = threadID / (objectDimension * objectDimension);
		pictureOffset = getPictureOffset(matchingOffset, objectOffset, pictureDimension, objectDimension);	
	
		// Calculates difference value and adds it to the right matching
		atomicAdd(&(deviceMatchingsArray[matchingOffset]), difference(devicePictureColorsMatrix[pictureOffset], deviceObjectColorsMatrix[objectOffset]));
		__syncthreads();
		
		// Converts calculated matching values into flags
		if (objectOffset == 0)
			devicePositionFlagsArray[matchingOffset] = (deviceMatchingsArray[matchingOffset] != 0) && (deviceMatchingsArray[matchingOffset] < matchingEpsilon);
		__syncthreads();		
	}
}

// ======================================= Entry Point =======================================
__host__ int* searchOnGPU(int pictureDimension, int *devicePictureColorsMatrix, Image object, float matchingEpsilon) 
{
	int positionsPerDimension = getPositionsPerDimension(pictureDimension, object.dimension), positionsCount = pow(positionsPerDimension, 2);
	int differencesPerDimension = positionsPerDimension * object.dimension;
	int blocksPerDimension = (differencesPerDimension / BLOCKDIMENSION) + ((differencesPerDimension / BLOCKDIMENSION) != 0);  
	int *hostPositionFlagsArray, *devicePositionFlagsArray, *deviceObjectColorsMatrix;
	float *deviceMatchingsArray;

	dim3 gridDimensions(blocksPerDimension, blocksPerDimension);
	dim3 blockDimensions(BLOCKDIMENSION, BLOCKDIMENSION);
	hipError_t error = hipSuccess;
		
	// Allocates memory for the position flags array
	hostPositionFlagsArray = (int*)malloc(positionsCount * sizeof(int));
	if (hostPositionFlagsArray == NULL) 
	{
		printf("Cannot allocate meory for position flags array\n");
		exit(0);
	}
	
	// Allocates and initializes required variables on the GPU
	allocateImageOnGPU(object, &deviceObjectColorsMatrix);
	
	error = hipMalloc(&devicePositionFlagsArray, positionsCount * sizeof(int));
	if (error != hipSuccess)
	{
		printf("Cannot allocate GPU memory for position flags array: %s (%d)\n", hipGetErrorString(error), error);
    		exit(0);
  	}
  	
  	error = hipMalloc(&deviceMatchingsArray, positionsCount * sizeof(float));
	if (error != hipSuccess)
	{
		printf("Cannot allocate GPU memory for matchings array: %s (%d)\n", hipGetErrorString(error), error);
    		exit(0);
  	}
  	
  	error = hipMemset(deviceMatchingsArray, 0, positionsCount * sizeof(float));
	if (error != hipSuccess)
	{
		printf("Cannot initialize matchings array on GPU: %s (%d)\n", hipGetErrorString(error), error);
    		exit(0);
  	}
  	
  	// Searches the object in the picture using CUDA - each block searches 1024 positions in the picture
  	searchPositions<<<gridDimensions, blockDimensions>>>(pictureDimension, devicePictureColorsMatrix,
  		object.dimension, deviceObjectColorsMatrix, deviceMatchingsArray, devicePositionFlagsArray, matchingEpsilon);
	
	// Copies the position flags array from GPU to host
	error = hipMemcpy(hostPositionFlagsArray, devicePositionFlagsArray, positionsCount * sizeof(int), hipMemcpyDeviceToHost);
	if (error != hipSuccess)
	{
		printf("Cannot copy position flags from GPU to host: %s (%d)\n", hipGetErrorString(error), error);
    		exit(0);
  	}
	
	// Frees allocated variables from the GPU
	error = hipFree(deviceMatchingsArray);
	if (error != hipSuccess)
	{
		printf("Cannot free matchings array from GPU: %s (%d)\n", hipGetErrorString(error), error);
    		exit(0);
  	}
	
	error = hipFree(devicePositionFlagsArray);
	if (error != hipSuccess)
	{
		printf("Cannot free position flags array from GPU: %s (%d)\n", hipGetErrorString(error), error);
    		exit(0);
  	}
	
	freeImageFromGPU(&deviceObjectColorsMatrix);
	
	return hostPositionFlagsArray;
}
